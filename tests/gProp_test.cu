#include <fftw3.h>
#include <complex>
#include <stdio.h>
#include <memory>
#include "openol.h"
#include "para.h"
#include <chrono>

#ifndef PROJECT_ROOT
#define PROJECT_ROOT
#endif

// #define WIDTH 512
// #define HEIGHT 512
// #define RECT_TEST


// typedef float PREC_T;

typedef std::complex<PREC_T> Complex;

int main(){
    

    auto img = std::make_unique<uint8_t[]>(HEIGHT * WIDTH);
    
    
    // std::complex<PREC_T> *u1;
    // hipHostMalloc(&u1,WIDTH * HEIGHT * sizeof(std::complex<PREC_T>),hipHostRegisterDefault);
    // std::unique_ptr<std::complex<PREC_T>[]> u(u1);
    auto u = std::make_unique<std::complex<PREC_T>[]>(WIDTH * HEIGHT);
    // hipHostRegister(u.get(),WIDTH * HEIGHT * sizeof(std::complex<PREC_T>),hipHostRegisterDefault);
    

    #ifndef RECT_TEST
    ol::bmpread(path,img.get(),HEIGHT,WIDTH);
    ol::img2complex(img,u,HEIGHT,WIDTH,true);
    #endif

    #ifdef RECT_TEST
    int rectwidth = 10;
    int rectheight = 10;
    for(int i = 0;i < HEIGHT;i++){
		for(int j = 0;j < WIDTH;j++){
			int adr = j + i * WIDTH;
			if(j > (WIDTH - rectwidth)/2 && j < (WIDTH + rectwidth)/2 && i > (HEIGHT-rectheight)/2 && i < (HEIGHT+rectheight)/2 ){
				u[adr]=1.0;
			}
			else{
				u[adr]=0.0;
			}
		}
	}
    #endif

    auto d_u = cuda::make_unique<thrust::complex<PREC_T>[]>(HEIGHT * WIDTH);
    std::chrono::system_clock::time_point  start, end;
    start = std::chrono::system_clock::now();
    ol::cpu2cuda(u,d_u,HEIGHT * WIDTH);
    
    // ol::Prop(d_u,HEIGHT,WIDTH,p,p,lambda,-d-0.000f,ol::FRESNEL);
    // ol::gFresnelPropBandLimit(d_u,HEIGHT,WIDTH,p,p,lambda,-d-0.000f);
    // ol::shiftedProp(d_u,d_u,HEIGHT,WIDTH,p,p,lambda,-d,-HEIGHT * p * 0.5f,-WIDTH * p * 0.5f,ol::PROPMODE::FRESNEL);
    // ol::gsplitProp(u,HEIGHT,WIDTH,p,p,lambda,-d,ol::PROPMODE::ASM);
    
    // ol::gshiftedFresnelProp(d_u,d_u,HEIGHT,WIDTH,(PREC_T)p/s,(PREC_T)p/s,(PREC_T)lambda,(PREC_T)-d,(PREC_T)s,-HEIGHT * p * 0.0f,-WIDTH * p * 0.0f);
    ol::ARSSFresnelProp(d_u,d_u,HEIGHT,WIDTH,(PREC_T)p/s,(PREC_T)p/s,(PREC_T)lambda,(PREC_T)-d,(PREC_T)s);
    ol::cuda2cpu(d_u,u,HEIGHT*WIDTH);

    end = std::chrono::system_clock::now();  // 計測終了時間
    auto msec = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << msec << " msec" << std::endl;
    ol::Save(PROJECT_ROOT "/out/olProp.bmp",u,HEIGHT,WIDTH,ol::AMP);
}